#include "hip/hip_runtime.h"

#include <>
#include <stdio.h>

__forceinline__ __device__ float3 get(uchar3* src, int x, int y, int w, int h) {
	if (x < 0 || x >= w || y < 0 || y >= h) return make_float3(0.5, 0.5, 0.5);
	uchar3 temp = src[y*w + x];
	return make_float3(float(temp.x) / 255., float(temp.y) / 255., float(temp.z) / 255.);
}
__global__ void resizeNormKernel(uchar3* src, float *dst, int dstW, int dstH, int srcW, int srcH,
	float scaleX, float scaleY, float shiftX, float shiftY) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int x = idx % dstW;
	const int y = idx / dstW;
	if (x >= dstW || y >= dstH)
		return;
	float w = (x - shiftX + 0.5) * scaleX - 0.5;
	float h = (y - shiftY + 0.5) * scaleY - 0.5;
	int h_low = (int)h;
	int w_low = (int)w;
	int h_high = h_low + 1;
	int w_high = w_low + 1;
	float lh = h - h_low;
	float lw = w - w_low;
	float hh = 1 - lh, hw = 1 - lw;
	float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
	float3 v1 = get(src, w_low, h_low, srcW, srcH);
	float3 v2 = get(src, w_high, h_low, srcW, srcH);
	float3 v3 = get(src, w_low, h_high, srcW, srcH);
	float3 v4 = get(src, w_high, h_high, srcW, srcH);
	int stride = dstW * dstH;
	dst[y*dstW + x] = w1 * v1.x + w2 * v2.x + w3 * v3.x + w4 * v4.x;
	dst[stride + y * dstW + x] = w1 * v1.y + w2 * v2.y + w3 * v3.y + w4 * v4.y;
	dst[stride * 2 + y * dstW + x] = w1 * v1.z + w2 * v2.z + w3 * v3.z + w4 * v4.z;
}
__global__ void myresizeNormKernel(uchar3* src, float *dst, int dstW, int dstH, int srcW, int srcH,
	float scaleX, float scaleY, float shiftX, float shiftY) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;//cuda线程的x索引.x对应的comuln.这里的x y是相对输出来说的,是指输出的宽高,不是输入的.
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= dstW || y >= dstH)
		return;
	float w = (x - shiftX + 0.5) * scaleX - 0.5;
	float h = (y - shiftY + 0.5) * scaleY - 0.5;
	int h_low = (int)h;
	int w_low = (int)w;
	int h_high = h_low + 1;
	int w_high = w_low + 1;
	float lh = h - h_low;
	float lw = w - w_low;
	float hh = 1 - lh, hw = 1 - lw;
	float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
	float3 v1 = get(src, w_low, h_low, srcW, srcH);
	float3 v2 = get(src, w_high, h_low, srcW, srcH);
	float3 v3 = get(src, w_low, h_high, srcW, srcH);
	float3 v4 = get(src, w_high, h_high, srcW, srcH);
	int stride = dstW * dstH;
	dst[y*dstW + x] = w1 * v1.x + w2 * v2.x + w3 * v3.x + w4 * v4.x;
	dst[stride + y * dstW + x] = w1 * v1.y + w2 * v2.y + w3 * v3.y + w4 * v4.y;
	dst[stride * 2 + y * dstW + x] = w1 * v1.z + w2 * v2.z + w3 * v3.z + w4 * v4.z;
}
int resizeAndNorm(void * p, float *d, int w, int h, int in_w, int in_h, hipStream_t stream) {
	float scaleX = (w*1.0f / in_w);
	float scaleY = (h*1.0f / in_h);
	float shiftX = 0.f, shiftY = 0.f;
	const int n = in_w * in_h;
	int blockSize = 1024;
	const int gridSize = (n + blockSize - 1) / blockSize;
	resizeNormKernel << <gridSize, blockSize, 0, stream >> > ((uchar3*)(p), d, in_w, in_h, w, h, scaleX, scaleY, shiftX, shiftY);
	return 0;
}
int myresizeAndNorm(void * p, float *d, int w, int h, int in_w, int in_h, hipStream_t stream) {
	float scaleX = (w*1.0f / in_w);
	float scaleY = (h*1.0f / in_h);
	float shiftX = 0.f, shiftY = 0.f;
	const int n = in_w * in_h;
	const dim3 blockDim(32,32);
	const dim3 gridDim((in_w+32-1)/32,(in_h+32-1)/32);
	int blockSize = 1024;
	const int gridSize = (n + blockSize - 1) / blockSize;
	myresizeNormKernel << <gridDim, blockDim, 0, stream >> > ((uchar3*)(p), d, in_w, in_h, w, h, scaleX, scaleY, shiftX, shiftY);
	return 0;
}