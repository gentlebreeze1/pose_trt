#include "hip/hip_runtime.h"
#include <cstring>
#include <vector>
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"

#include "NvInfer.h"
#include "NvInferPlugin.h"

#include "plugin_utils.h"
#include "PReLUPlugin.h"

#include "spdlog/spdlog.h"

static const char* G_PRELU_TYPE = "PReLU";
static const char* G_PRELU_NAME = "PReLU_TRT"; //plugin_name = plugin_type + plugin_namespace

// CUDA: use 512 threads per block
static const int CUDA_NUM_THREADS = 512;

// CUDA: number of blocks for threads.
inline int CAFFE_GET_BLOCKS(const int N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

// /******** PReLU CUDA function ********/
// CUDA kernele for forward
template <typename Ftype>
__global__ void PReLUForward(const int n, const int channels, const int dim,
    const Ftype* slope_data,
    const Ftype* in, Ftype* out,
    const Ftype zero,
    const int div_factor) {
    CUDA_KERNEL_LOOP(index, n) {
        int c = (index / dim) % channels / div_factor;
        if(in[index] > zero) {
            out[index] = in[index];
        } else {
            out[index] = in[index] * slope_data[c];
        }
    }
}
template <typename Ftype>
hipError_t Forward_gpu(const int count, const int channels, const int dim,
                const Ftype* mDeviceKernel,
                const Ftype* bottom_data, Ftype* top_data, 
                const Ftype zero,
                const int div_factor, const hipStream_t stream) {
    PReLUForward<<<CAFFE_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>
        (count, channels, dim, mDeviceKernel, bottom_data, top_data, zero, div_factor);
    hipError_t err = hipGetLastError();
    return err;
}

PReLUPlugin::PReLUPlugin(const nvinfer1::Weights *weights, int nbWeights) {
    mWeights = weights[0];
    mWeights.values = malloc(mWeights.count * type2size(mWeights.type));
    memcpy(const_cast<void *>(mWeights.values), weights[0].values, mWeights.count * type2size(mWeights.type));
}

// create the plugin at runtime from a byte stream
PReLUPlugin::PReLUPlugin(const void *data, size_t length) {
    const char *d = static_cast<const char *>(data), *a = d;
    read<int>(d, mNbInputChannels);
    read<int>(d, mNbInputHeight);
    read<int>(d, mNbInputWidth);
    read<nvinfer1::DataType>(d, mDataType);
    read<int64_t>(d, mWeights.count);
    read<nvinfer1::DataType>(d, mWeights.type);
    mWeights.values = nullptr;
    mWeights.values = malloc(mWeights.count * type2size(mWeights.type));
    memcpy(const_cast<void *>(mWeights.values), d, mWeights.count * type2size(mWeights.type));
    d = d + mWeights.count * type2size(mWeights.type);
    ASSERT(d == a + length);
}

size_t PReLUPlugin::getSerializationSize() const {
    return sizeof(mNbInputChannels) + sizeof(mNbInputWidth) + sizeof(mNbInputHeight) + sizeof(mDataType) + 
           sizeof(mWeights.count) + sizeof(mWeights.type) + mWeights.count * type2size(mWeights.type);
}

void PReLUPlugin::serialize(void *buffer) const {
    char *d = static_cast<char *>(buffer), *a = d;
    write(d, mNbInputChannels);
    write(d, mNbInputHeight);
    write(d, mNbInputWidth);
    write(d, mDataType);
    write(d, mWeights.count);
    write(d, mWeights.type);
    convertAndCopyToBuffer(d, mWeights, mWeights.type);
    ASSERT(d == a + getSerializationSize());
}

PReLUPlugin::~PReLUPlugin() {
    if (mWeights.values) 
    {
        free(const_cast<void *>(mWeights.values));
        mWeights.values = nullptr;
    }
    if (mDeviceKernel) 
    {
        hipFree(mDeviceKernel);
        mDeviceKernel = nullptr;
    }
}

int PReLUPlugin::getNbOutputs() const {
    return 1;
}

nvinfer1::Dims PReLUPlugin::getOutputDimensions(int index, const nvinfer1::Dims* inputs, int nbInputDims) {
    if(index == 0) {
        return nvinfer1::Dims3(inputs[0].d[0],inputs[0].d[1],inputs[0].d[2]);
    } // else if(index == n) {
        // for other outputs if exists.
    // }
    else {
        ASSERT(false);
    }
}

bool PReLUPlugin::supportsFormat(nvinfer1::DataType type, nvinfer1::PluginFormat format) const {
    return (type == nvinfer1::DataType::kFLOAT | type == nvinfer1::DataType::kHALF) 
            && format == nvinfer1::PluginFormat::kNCHW;
}

void PReLUPlugin::configureWithFormat(const nvinfer1::Dims* inputDims, int nbInputs, 
                                      const nvinfer1::Dims* outputDims, int nbOutputs,
                                      nvinfer1::DataType type, nvinfer1::PluginFormat format, 
                                      int maxBatchSize) {
    ASSERT((type == nvinfer1::DataType::kFLOAT | type == nvinfer1::DataType::kHALF)
            && format == nvinfer1::PluginFormat::kNCHW);
    mNbInputChannels = inputDims[0].d[0]; 
    mNbInputHeight = inputDims[0].d[1];
    mNbInputWidth = inputDims[0].d[2];
    mDataType = type;
}

int PReLUPlugin::initialize() {
    convertAndCopyToDeivce(mDeviceKernel, mWeights, mDataType);
    return 0;
}

void PReLUPlugin::terminate() {
    if (mWeights.values)
    {
        free(const_cast<void *>(mWeights.values));
        mWeights.values = nullptr;
    }
    if (mDeviceKernel)
    {
        hipFree(mDeviceKernel);
        mDeviceKernel = nullptr;
    }
}

size_t PReLUPlugin::getWorkspaceSize(int maxBatchSize) const
{
    return 0;
}

int PReLUPlugin::enqueue(int batchSize, const void *const *inputs, void **outputs, void *workspace, hipStream_t stream)
{
    const int count = batchSize * mNbInputChannels * mNbInputWidth * mNbInputHeight;
    const int channels = mNbInputChannels;
    const int dim = mNbInputWidth * mNbInputHeight;
    const int div_factor = 1;
    if (mDataType == nvinfer1::DataType::kFLOAT)
    {
        const float zerof{0.0f};
        CUDA_CHECK(Forward_gpu(count, channels, dim,
                            reinterpret_cast<const float *>(mDeviceKernel),
                            reinterpret_cast<const float *>(inputs[0]),
                            reinterpret_cast<float *>(outputs[0]),
                            zerof,
                            div_factor,
                            stream));
    } else {
        const __half zeroh = __half(0.0f);
        CUDA_CHECK(Forward_gpu(count, channels, dim,
                            reinterpret_cast<const __half *>(mDeviceKernel),
                            reinterpret_cast<const __half *>(inputs[0]),
                            reinterpret_cast<__half *>(outputs[0]),
                            zeroh,
                            div_factor,
                            stream));
    }

    return 0;
}

const char *PReLUPlugin::getPluginType() const {
    return G_PRELU_TYPE;
}

const char *PReLUPlugin::getPluginVersion() const {
    return G_PLUGIN_VERSION;
}

void PReLUPlugin::destroy() {
    delete this; 
}

nvinfer1::IPluginV2* PReLUPlugin::clone() const {
    return new PReLUPlugin(&mWeights, 1);
}

const char* PReLUPlugin::getPluginNamespace() const {
    return G_PLUGIN_NAMESPACE;
}

PReLUPluginCreator::PReLUPluginCreator()  {
    mPluginAttributes.emplace_back(nvinfer1::PluginField("weights", nullptr, nvinfer1::PluginFieldType::kFLOAT32, 1));
    mPluginAttributes.emplace_back(nvinfer1::PluginField("nbWeight", nullptr, nvinfer1::PluginFieldType::kINT32, 1));
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

// return PRELU_PLUGIN_TYPE + PRELU_PLUGIN_NAMESPACE
const char* PReLUPluginCreator::getPluginName() const {
    // std::string plugin_type{G_PRELU_TYPE};
    // std::string plugin_namespace{G_PLUGIN_NAMESPACE};
    // return (plugin_type+plugin_namespace).c_str();
    return G_PRELU_NAME;
}

const char* PReLUPluginCreator::getPluginVersion() const {
    return G_PLUGIN_VERSION;
}

const nvinfer1::PluginFieldCollection* PReLUPluginCreator::getFieldNames() {
    return &mFC;
}

nvinfer1::IPluginV2* PReLUPluginCreator::createPlugin(const char* name, const nvinfer1::PluginFieldCollection* fc) {
    int nbWeights;
    std::vector<float> weightValues;
    const nvinfer1::PluginField* fields = fc->fields;
    for (int i=0; i<fc->nbFields; i++) {
        const char* attrName = fields[i].name;
        if(strcmp(attrName, "nbWeights")) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kINT32);
            nbWeights = *(static_cast<const int*>(fields[i].data));
        }
        if(strcmp(attrName, "weights")) {
            ASSERT(fields[i].type == nvinfer1::PluginFieldType::kFLOAT32);
            weightValues.reserve(fields[i].length);
            const auto* w = static_cast<const float*>(fields[i].data);
            for (int j = 0; j < weightValues.size(); j++)
            {
                weightValues.push_back(*w);
                w++;
            }
        }
    }
    nvinfer1::Weights weights{nvinfer1::DataType::kFLOAT, weightValues.data(), (int64_t)weightValues.size()};
    return new PReLUPlugin(&weights,nbWeights);
}

// deserialization plugin implementation
nvinfer1::IPluginV2* PReLUPluginCreator::deserializePlugin(const char *layerName, const void *serialData, size_t serialLength) {
    return new PReLUPlugin(serialData, serialLength);
}

const char* PReLUPluginCreator::getPluginNamespace() const {
    return G_PLUGIN_NAMESPACE;
}

REGISTER_TENSORRT_PLUGIN(PReLUPluginCreator); // DO NOT FORGET THIS
                                              // 别忘了这个
