#include "hip/hip_runtime.h"
#include "decode.h"
#include "Trt.h"
#include "utils.h"
#include "math.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include "../plugin/YoloLayerPlugin/YoloLayerPlugin.hpp"
__device__ float Logist1(float data){ return 1./(1. + exp(-data)); };
__global__ void caldetection(const float* input,float* output,int noelements,int yolowidth,int yoloheight,
                             const float anchors[6],int classes,int outputElem)
{
//    printf("进入");
    int idx=threadIdx.x+blockDim.x*blockIdx.x;
    if(idx>noelements)return;
    int stride=yoloheight*yolowidth;
    int bnidx=idx/stride;
//    printf("input %f next is %f\n",input[0],input[1]);
    int curidx=idx-bnidx*stride;

//    printf("IDX is %d,bnidx is %d,curid is %d\n",idx,bnidx,curidx);
    const float* curinput=input+bnidx*(7)*stride*3;
    for(int k=0;k<3;k++)
    {
        int beginidx=(7*stride)*k+curidx;
        int objidx=beginidx+stride*4;
        float objprob=Logist1(curinput[objidx]);
//        ofstream out_txt;
//        out_txt.open("./22.txt",ios::app);
//
//        out_txt<<curidx<<" "<<beginidx<<" "<<curinput[objidx]<<" "<<objprob<<"\n";
//        printf("curidx is%d, begin id is%d, input is %f 得分是%f\n",curidx,beginidx,curinput[objidx],objprob);
        if(objprob <= 0.7)
            continue;
//        printf("走到这一步\n");
        int row = curidx / yolowidth;
        int cols = curidx % yolowidth;

        //classes
        int classId = -1;
        float maxProb = IGNORE_THRESH;
        for (int c = 0;c<2;++c){
            float cProb =  Logist1(curinput[beginidx + (5 + c) * stride]) * objprob;
            if(cProb > maxProb){
                maxProb = cProb;
                classId = c;
            }
        }
        if(classId >= 0) {
            float *curOutput = output + bnidx*outputElem;
            int resCount = (int)atomicAdd(curOutput,1);
            char* data = (char * )curOutput + sizeof(float) + resCount*sizeof(Detection);
            Detection* det =  (Detection*)(data);

            //Location
            det->bbox[0] = (cols + Logist1(curinput[beginidx]))/ yolowidth;
            det->bbox[1] = (row + Logist1(curinput[beginidx+stride]))/ yoloheight;
            det->bbox[2] = exp(curinput[beginidx+2*stride]) * anchors[2*k];
            det->bbox[3] = exp(curinput[beginidx+3*stride]) * anchors[2*k + 1];
//            printf("box value is%f ,%f,%f,%f",det->bbox[0],det->bbox[1],det->bbox[2],det->bbox[3]);
            float tem_cla=float(classId);
            det->classId = llround(double(tem_cla));
            det->prob = maxProb;
//            printf("det x is%f y is%f w is%f h is%f \n",det->bbox[0],det->bbox[1],det->bbox[2],det->bbox[3]);
//            printf("class id is %d,temp is %f score is %f\n",classId,tem_cla,maxProb);
        }
//        out_txt.close();
    }

}
hipError_t decode_gpu(vector<float> input,YoloKernel yolo_kernel,vector<Detection>& output)
{
    float* temp_input;
    int input_num=input.size();
    CUDA_CHECK(hipMalloc(&temp_input,input.size()*sizeof(float)));
    CUDA_CHECK(hipMemcpy(temp_input,&input[0],input.size()*sizeof(float),hipMemcpyHostToDevice));
    float* output1;
    void* devAnchor;
    size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
    CUDA_CHECK(hipMalloc(&devAnchor,AnchorLen));
    int outputElem = 1;
    outputElem+=yolo_kernel.width*yolo_kernel.height*3*sizeof(Detection)/sizeof(float);
    CUDA_CHECK(hipMalloc(&output1,sizeof(float)*outputElem));

    int numelem=yolo_kernel.width*yolo_kernel.height;
    CUDA_CHECK(hipMemcpyAsync(devAnchor,yolo_kernel.anchors,AnchorLen,hipMemcpyHostToDevice));
    caldetection<<<(yolo_kernel.width*yolo_kernel.height+512-1)/512,512>>>
    (temp_input,output1,numelem,yolo_kernel.width,yolo_kernel.height,(float *)devAnchor,2,outputElem);

    hipError_t cudaStatus;
    hipFree(devAnchor);
    float* out_host{};
    CUDA_CHECK(hipHostMalloc(&out_host,sizeof(float)*outputElem));

    CUDA_CHECK(hipMemcpy(out_host,output1,sizeof(float)*outputElem,hipMemcpyDeviceToHost));
//    printf("第一个输出%f,第二个输出%f",out_host[0],out_host[1]);
    hipFree(output1);
    for(int k=0;k<int(out_host[0]);k++)
    {
        Detection temp;
        temp.bbox[0]=out_host[6*k+1];
        temp.bbox[1]=out_host[6*k+2];
        temp.bbox[2]=out_host[6*k+3];
        temp.bbox[3]=out_host[6*k+4];
        temp.classId=out_host[6*k+5];
        temp.prob=out_host[6*k+6];
        output.push_back(temp);
//        printf("det x is %f,det y is %f,det w is %f,det h is %f,det cla is %d det scor is %f\n"
//                ,out_host[6*k+1],out_host[6*k+2],out_host[6*k+3],out_host[6*k+4],int(out_host[6*k+5]),out_host[6*k+6]);

    }
//
    hipHostFree(out_host);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error " << hipGetErrorString(cudaStatus) << " at " << __FILE__ << ":" << __LINE__ << std::endl;


    }



//    CUDA_CHECK(hipFree(devAnchor));

    return hipGetLastError();
}